#include "hip/hip_runtime.h"
#include "../include/scalar_theory_gpu/scalar_theory_gpu_helper.hpp"


dev_vec gen_dev_vec(const uint n_sites)
{
    return dev_vec(n_sites, 0.0);
}


dev_vec_vec_int set_nearest_neighbours(thrust::host_vector< thrust::host_vector<int> > neighbour_indices)
{
    dev_vec_vec_int neighbour_indices_dev;
    for(auto& nn_of_site : neighbour_indices) {
        dev_vec_int nn_of_site_dev = nn_of_site;
        neighbour_indices_dev.push_back(nn_of_site_dev);
    }
    return std::move(neighbour_indices_dev);
}


double site_helper(const dev_vec &lattice, int i)
{
    return lattice[i];
}


void initialize_helper(std::string starting_mode, dev_vec &lattice, thrust::device_vector<hiprandState> &s,
                       unsigned long long &rnd_offset, const double epsilon)
{
    if(starting_mode == "hot") {
        s = thrust::device_vector<hiprandState>(lattice.size());

        // Initialize the random generators
        thrust::for_each_n(thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0), s.begin())),
                           lattice.size(), curand_setup(rnd_offset));

        // Initialize each lattice site with a random Gaussian number, where 2 * epsilon refers to the standard deviation
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(s.begin(), lattice.begin())),
                         thrust::make_zip_iterator(thrust::make_tuple(s.end(), lattice.end())),
                         initialization(epsilon));
        rnd_offset += 1;
    }
    else
        thrust::fill(lattice.begin(), lattice.end(), 0.0);
    // print_range("Initialized Lattice", lattice.begin(), lattice.end());
}


void update_step_helper(uint measure_interval, dev_vec &lattice, dev_vec &current_lattice, dev_vec &momenta,
                        dev_vec &current_momenta, dev_vec &drift, dev_vec &identity_scalar_product,
                        thrust::device_vector<hiprandState> &s, dev_vec_vec_int &neighbour_indices,
                        std::uniform_real_distribution<double> &rand, unsigned long long &rnd_offset, const double dt,
                        const int n, const double m, const double kappa, const double lambda)
{
    for(uint i = 0; i < measure_interval; i++)
    {
        auto current_action = action_helper(lattice, drift, identity_scalar_product, neighbour_indices, kappa, lambda);
        thrust::copy(lattice.begin(), lattice.end(), current_lattice.begin());

        // Sample momenta
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(s.begin(), momenta.begin())),
                         thrust::make_zip_iterator(thrust::make_tuple(s.end(), momenta.end())),
                         momentum_initialization());
        // Update offset for random number generation
        rnd_offset += 1;

        thrust::copy(momenta.begin(), momenta.end(), current_momenta.begin());

        // Hamiltonians' equation - Leapfrog
        for(auto i = 0; i < n; i++)
        {
            drift_computation(lattice, drift, identity_scalar_product, neighbour_indices, rnd_offset, kappa, lambda);
            thrust::transform(momenta.begin(), momenta.end(), drift.begin(), momenta.begin(), leapfrog_momenta_update(dt));

            thrust::transform(lattice.begin(), lattice.end(), momenta.begin(), lattice.begin(), leapfrog_lattice_update(dt, m));

            drift_computation(lattice, drift, identity_scalar_product, neighbour_indices, rnd_offset, kappa, lambda);
            thrust::transform(momenta.begin(), momenta.end(), drift.begin(), momenta.begin(), leapfrog_momenta_update(dt));
        }

        auto proposal_action = action_helper(lattice, drift, identity_scalar_product, neighbour_indices, kappa, lambda);

        auto current_kinetic_term = thrust::inner_product(current_momenta.begin(), current_momenta.end(), current_momenta.begin(), 0.0);
        auto proposal_kinetic_term = thrust::inner_product(momenta.begin(), momenta.end(), momenta.begin(), 0.0);

        // std::cout << proposal_action + 0.5 / m * proposal_kinetic_term << " == " << current_action + 0.5 / m * current_kinetic_term << std::endl;

        // Accept/Reject step
        if (rand(mcmc::util::g_gen) >= std::min(1.0, std::exp(
                -1.0 * (proposal_action - current_action) - 0.5 * (proposal_kinetic_term - current_kinetic_term) / m))) {
            thrust::copy(current_lattice.begin(), current_lattice.end(), lattice.begin()); // Reject
        }
        // else{} // Accept
    }
}


// Computes the scalar product separately for each lattice site
void compute_square(const dev_vec &lattice, dev_vec &identity_scalar_product)
{
    thrust::transform(lattice.begin(), lattice.end(), identity_scalar_product.begin(), square());
}


void drift_computation(const dev_vec &lattice, dev_vec &drift, dev_vec &identity_scalar_product,
                       const dev_vec_vec_int& neighbour_indices, unsigned long long &rnd_offset,
                       const double kappa, const double lambda)
{
    thrust::fill(drift.begin(), drift.end(), 0.0);

    // Computes for each phi_i the drift term:
    //     -2.0 * kappa \sum_neighbours \phi_j + 2.0 * (1 - 2.0 * lambda) phi_i + 4.0 * lambda * phi_i^3,
    // where the sum runs over all neighbours of phi_i and performs a Langevin update.

    //[ Compute interaction term \sum_neighbours \phi_neighbour

    // Sum over neighbours for each site AND each component by looping over all neighbours
    for(size_t n = 0; n < neighbour_indices.size(); n++) {
        thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                        drift.begin(), thrust::make_permutation_iterator(lattice.begin(), neighbour_indices[n].begin()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                        drift.end(), thrust::make_permutation_iterator(lattice.end(), neighbour_indices[n].end()))),
                drift_interaction()
        );
    }
    //]

    //[ Compute the scalar product \phi^2

    // Compute the inner product
    compute_square(lattice, identity_scalar_product);

    //[ Add remaining terms and finalize the drift computation
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(lattice.begin(), drift.begin(), identity_scalar_product.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(lattice.end(), drift.end(), identity_scalar_product.end())),
                     compute_drift(kappa, lambda));
    // Update offset for random number generation
    rnd_offset += 1;
}


double action_helper(const dev_vec &lattice, dev_vec& potential, dev_vec &identity_scalar_product,
                     const dev_vec_vec_int& neighbour_indices, const double kappa, const double lambda)
{
    // Computes the action of the scalar theory according to
    //     S = -2.0 * kappa \sum_<i, j> \phi_i \phi_j + \sum_i (1 - 2.0 * lambda) phi_i^2 + lambda * phi_i^4
    // where the sum runs over neighbour pairs phi_i (note that in the computation only the right hand side neighbours
    // need to be taken into account.

    thrust::fill(potential.begin(), potential.end(), 0.0);

    //[ Compute interaction term \sum_neighbours \phi * \phi_neighbour

    // Sum over neighbours for each site AND each component
    for(size_t n = 0; n < neighbour_indices.size(); n +=2) // only positive neighbours!!
    {
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(
                potential.begin(), lattice.begin(), thrust::make_permutation_iterator(lattice.begin(), neighbour_indices[n].begin()))),
                         thrust::make_zip_iterator(thrust::make_tuple(
                                 potential.end(), lattice.end(), thrust::make_permutation_iterator(lattice.end(), neighbour_indices[n].end()))),
                         action_interaction());
    }
    //]

    //[ Compute the scalar product \phi^2
    compute_square(lattice, identity_scalar_product);
    //]

    //[ Add remaining terms and compute sum

    // <-> potential = -2.0 * mp.kappa * potential + site * site + mp.lambda * pow(site * site - 1.0, 2.0);
    thrust::transform(potential.begin(), potential.end(), identity_scalar_product.begin(), potential.begin(), compute_action(kappa, lambda));
    return thrust::reduce(potential.begin(), potential.end());
    //]
}


// Accumulates all sites of the lattice
double accumulate(const dev_vec &lattice)
{
    return thrust::reduce(lattice.begin(), lattice.end());
}


// Second moment
double second_moment_helper(const dev_vec &lattice, dev_vec &identity_scalar_product)
{
    compute_square(lattice, identity_scalar_product);
    return thrust::reduce(identity_scalar_product.begin(), identity_scalar_product.end()) / lattice.size();
}


// Fourth moment
double fourth_moment_helper(const dev_vec &lattice, dev_vec &identity_scalar_product)
{
    compute_square(lattice, identity_scalar_product);
    return thrust::inner_product(identity_scalar_product.begin(), identity_scalar_product.end(), identity_scalar_product.begin(), 0.0f) / lattice.size();
}


//* Implementation of a langevin update *//

/* void update_step_helper(uint measure_interval, dev_vec &lattice, thrust::device_vector<hiprandState> &s, dev_vec_vec_int &neighbour_indices,
                        unsigned long long &rnd_offset, const double epsilon, const double kappa, const double lambda)
{
    for(uint i = 0; i < measure_interval; i++)
    {
        // Computes for each phi_i the drift term:
        //     -2.0 * kappa \sum_neighbours \phi_j + 2.0 * (1 - 2.0 * lambda) phi_i + 4.0 * lambda * phi_i^3,
        // where the sum runs over all neighbours of phi_i and performs a Langevin update.

        //[ Compute interaction term \sum_neighbours \phi_neighbour

        dev_vec interaction_term(lattice.size(), 0.0);

        // Sum over neighbours for each site AND each component by looping over all neighbours
        for(size_t n = 0; n < neighbour_indices.size(); n++) {
            thrust::for_each(
                    thrust::make_zip_iterator(thrust::make_tuple(
                            interaction_term.begin(), thrust::make_permutation_iterator(lattice.begin(), neighbour_indices[n].begin()))),
                    thrust::make_zip_iterator(thrust::make_tuple(
                            interaction_term.end(), thrust::make_permutation_iterator(lattice.end(), neighbour_indices[n].end()))),
                    drift_interaction()
            );
        }
        //]

        //[ Compute the scalar product \phi^2

        // Compute the inner product
        dev_vec identity_scalar_product = compute_square(lattice);

        //[ Add remaining terms and update sites and add noise
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(s.begin(), lattice.begin(), interaction_term.begin(), identity_scalar_product.begin())),
                         thrust::make_zip_iterator(thrust::make_tuple(s.end(), lattice.end(), interaction_term.end(), identity_scalar_product.end())),
                         langevin_update(epsilon, kappa, lambda));
        // Update offset for random number generation
        rnd_offset += 1;
        //]
    }
} */