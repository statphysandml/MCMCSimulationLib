#include "../../include/thrust/thrust_header.hpp"

void print_system_info()
{
    int driver_version , runtime_version;
    hipDriverGetVersion( &driver_version );
    hipRuntimeGetVersion ( &runtime_version );
    std::cout << driver_version << "\t" << runtime_version << std::endl;
}