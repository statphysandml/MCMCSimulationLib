#include "hip/hip_runtime.h"
#include "../include/onmodel_gpu/onmodel_gpu_helper.hpp"


DevDatC gen_lattice(const uint N, const uint dim)
{
    return DevDatC(N, dim);
}


dev_vec_vec_int set_nearest_neighbours(thrust::host_vector< thrust::host_vector<int> > neighbour_indices)
{
    dev_vec_vec_int neighbour_indices_dev;
    for(auto& nn_of_site : neighbour_indices) {
        dev_vec_int nn_of_site_dev = nn_of_site;
        neighbour_indices_dev.push_back(nn_of_site_dev);
    }
    return std::move(neighbour_indices_dev);
}


std::vector<double> site_helper(const DevDatC &lattice, int i)
{
    return lattice.get_ith_element(i);
}


void initialize_helper(std::string starting_mode, uint N, DevDatC &lattice, thrust::device_vector<hiprandState> &s,
                       uint n_sites, unsigned long long &rnd_offset, const double epsilon)
{
    if(starting_mode == "hot") {
        s = thrust::device_vector<hiprandState>(lattice.size());

        rnd_offset = 0;

        // Initialize the random generators
        thrust::for_each_n(thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0), s.begin())),
                           lattice.size(), curand_setup(rnd_offset));

        // Initialize each lattice site with a random Gaussian number, where 2 * epsilon refers to the standard deviation
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(s.begin(), lattice.begin())),
                         thrust::make_zip_iterator(thrust::make_tuple(s.end(), lattice.end())),
                         initialization(epsilon));
        rnd_offset += 1;
    }
    else
        thrust::fill(lattice.begin(), lattice.end(), 0.0);
    // print_range("Initialized Lattice", lattice.begin(), lattice.end());
    // std::cout << " Expected lattice size " << n_sites * N << " = Actual lattice " << lattice.size() << std::endl;
}


void update_step_helper(uint measure_interval, uint N, DevDatC &lattice, thrust::device_vector<hiprandState> &s, dev_vec_vec_int &neighbour_indices,
                        uint n_sites, unsigned long long &rnd_offset, const double epsilon, const double kappa, const double lambda)
{
    for(uint i = 0; i < measure_interval; i++)
    {
        // Computes for each phi_i the drift term:
        //     -2.0 * kappa \sum_neighbours \phi_j + 2.0 * (1 - 2.0 * lambda) phi_i + 4.0 * lambda * phi_i^3,
        // where the sum runs over all neighbours of phi_i and performs a Langevin update.

        //[ Compute interaction term \sum_neighbours \phi_neighbour

        dev_vec interaction_term(n_sites * N, 0.0);

        // Sum over neighbours for each site AND each component by looping over all neighbours
        for(size_t n = 0; n < neighbour_indices.size(); n++) {
            thrust::for_each(
                    thrust::make_zip_iterator(thrust::make_tuple(
                            interaction_term.begin(), thrust::make_permutation_iterator(lattice.begin(), neighbour_indices[n].begin()))),
                    thrust::make_zip_iterator(thrust::make_tuple(
                            interaction_term.end(), thrust::make_permutation_iterator(lattice.end(), neighbour_indices[n].end()))),
                    drift_interaction()
            );
        }
        //]

        //[ Compute the scalar product \phi^2

        // Compute the inner product
        dev_vec identity_scalar_product = compute_inner_product(N, lattice, n_sites);

        // Tile to to have the scalar for each component
        tiled_range<dev_iterator> identity_scalar_product_tiled(
                identity_scalar_product.begin(), identity_scalar_product.end(), N);
        //]

        //[ Add remaining terms and update sites and add noise
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(s.begin(), lattice.begin(), interaction_term.begin(), identity_scalar_product_tiled.begin())),
                         thrust::make_zip_iterator(thrust::make_tuple(s.end(), lattice.end(), interaction_term.end(), identity_scalar_product_tiled.end())),
                         langevin_update(epsilon, kappa, lambda));
        // Update offset for random number generation
        rnd_offset += 1;
        //]
    }
}


// Computes the scalar product separately for each lattice site
dev_vec compute_inner_product(const uint N, const DevDatC &lattice, const uint n_sites)
{
    dev_vec identity_scalar_product(n_sites, 0.0);
    dev_vec identity_scalar_product_indices(n_sites);

    // Compute the square of each field an perform for each single field a sum over the components
    // keys = 0 0 0 0 1 1 1 1 2 2 2 2 ...
    // mask = 0 16 32 48 1 17 33 49 2 18 34 50 ... (as an example for a 4x4 lattice)
    thrust::reduce_by_key
            (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(N)),
             thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(N)) + (N * n_sites),
             thrust::make_permutation_iterator(
                     thrust::make_transform_iterator(lattice.begin(), square()),
                     thrust::make_transform_iterator(thrust::counting_iterator<int>(0), transpose<int>(N, n_sites))),
             identity_scalar_product_indices.begin(),
             identity_scalar_product.begin(),
             thrust::equal_to<int>(),
             thrust::plus<cudaT>());
    return std::move(identity_scalar_product);
}


double action_helper(const uint N, const DevDatC &lattice, const dev_vec_vec_int& neighbour_indices,
                     const uint n_sites, const double kappa, const double lambda)
{
    // Computes the action of the scalar theory according to
    //     S = -2.0 * kappa \sum_<i, j> \phi_i \phi_j + \sum_i (1 - 2.0 * lambda) phi_i^2 + lambda * phi_i^4
    // where the sum runs over neighbour pairs phi_i (note that in the computation only the right hand side neighbours
    // need to be taken into account.

    dev_vec potential(n_sites * N, 0.0);

    //[ Compute interaction term \sum_neighbours \phi * \phi_neighbour

    // Sum over neighbours for each site AND each component
    for(size_t n = 0; n < neighbour_indices.size(); n +=2) // only positive neighbours!!
    {
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(
                potential.begin(), lattice.begin(), thrust::make_permutation_iterator(lattice.begin(), neighbour_indices[n].begin()))),
                         thrust::make_zip_iterator(thrust::make_tuple(
                                 potential.end(), lattice.end(), thrust::make_permutation_iterator(lattice.end(), neighbour_indices[n].end()))),
                         action_interaction());
    }
    //]

    //[ Perform sum over components of interaction (Reduce from N * n_sites to n_sites by executing the some of the inner product \phi_i \phi_j)

    dev_vec potential_scalar_product_indices(n_sites);

    thrust::reduce_by_key
            // The first line does the same as https://github.com/NVIDIA/thrust/blob/main/examples/repeated_range.cu
            (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(N)),
             thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(N)) + (N * n_sites),
             thrust::make_permutation_iterator(
                     potential.begin(),
                     thrust::make_transform_iterator(thrust::counting_iterator<int>(0), transpose<int>(N, n_sites))),
             potential_scalar_product_indices.begin(),
             potential.begin(),
             thrust::equal_to<int>(),
             thrust::plus<cudaT>());

    potential.resize(n_sites);
    //]

    //[ Compute the scalar product \phi^2
    dev_vec identity_scalar_product = compute_inner_product(N, lattice, n_sites);
    //]

    //[ Add remaining terms and compute sum

    // <-> potential = -2.0 * mp.kappa * potential + site * site + mp.lambda * pow(site * site - 1.0, 2.0);
    thrust::transform(potential.begin(), potential.end(), identity_scalar_product.begin(), potential.begin(), compute_action(kappa, lambda));
    return thrust::reduce(potential.begin(), potential.end());
    //]
}


// Accumulates each component over the lattice
std::vector<double> accumulate(const uint N, const DevDatC &lattice, const uint n_sites)
{
    dev_vec sum(N, 0.0);
    dev_vec sum_indices(N);

    thrust::reduce_by_key
            // The first line does the same as https://github.com/NVIDIA/thrust/blob/main/examples/repeated_range.cu
            (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(n_sites)),
             thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(n_sites)) + (N * n_sites),
             lattice.begin(),
             sum_indices.begin(),
             sum.begin(),
             thrust::equal_to<int>(),
             thrust::plus<cudaT>());

    std::vector<double> sum_vec(N, 0);
    thrust::copy(sum.begin(), sum.end(), sum_vec.begin());
    return sum_vec;
}


// Second moment
double second_moment_helper(const uint N, const DevDatC &lattice, const uint n_sites)
{
    dev_vec identity_scalar_product = compute_inner_product(N, lattice, n_sites);
    return thrust::reduce(identity_scalar_product.begin(), identity_scalar_product.end()) / n_sites;
}


// Fourth moment
double fourth_moment_helper(const uint N, const DevDatC &lattice, const uint n_sites)
{
    dev_vec identity_scalar_product = compute_inner_product(N, lattice, n_sites);
    return thrust::inner_product(identity_scalar_product.begin(), identity_scalar_product.end(), identity_scalar_product.begin(), 0.0f) / n_sites;
}